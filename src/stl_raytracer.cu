#include "hip/hip_runtime.h"
#include <fstream>
#include <stdlib.h>
#include "shapes.hpp"
#include <string>
#include <iostream>
#include <cmath>
#include <time.h>


double CLOCK() {
        struct timespec t;
        clock_gettime(CLOCK_MONOTONIC,  &t);
        return (t.tv_sec * 1000)+(t.tv_nsec*1e-6);
}

#define MAX_PIXEL 1023
void clamp_pixels(Vec3& col)
{
  col.x = (col.x > MAX_PIXEL) ? MAX_PIXEL : (col.x < 0) ? 0 : col.x;
  col.y = (col.y > MAX_PIXEL) ? MAX_PIXEL : (col.y < 0) ? 0 : col.y;
  col.z = (col.z > MAX_PIXEL) ? MAX_PIXEL : (col.z < 0) ? 0 : col.z;
}



__device__ bool ray_triangle_intersect(struct Ray * ray, struct Triangle * tri, struct Vec3 * intersection_point){
  // error bound for 0
  const float epsilon = 0.0000001;

  struct Vec3 c_a_vector = tri->v2 - tri->v0; //edge2
  struct Vec3 b_a_vector = tri->v1 - tri->v0; //edge1 

  struct Vec3 *d_cross_c_a = cross_vec3(ray->d, c_a_vector); // h = ray cross edge2

  // first calculating determinant, 
  //  if its ~0 then the ray is parallel to the triangle
  //  if it is <0, then we are hitting the back of the triangle (counting as not intersecting for now)
  //  this will need to be adjusted in the future (especially with refraction) (i.e. use absolute value of det)
  // can therefore ignore it  
  double det = dot_vec3(*d_cross_c_a, b_a_vector); // a = (ray cross edge2) => h dot edge1
  if (det < epsilon && det > -epsilon){
    free(d_cross_c_a);
    return false;
  }
  double inv_det = 1.0 / det; // f = 1/a

  struct Vec3 o_a_vector = ray->o - tri->v0; // s = ray origin - vertex0

  // start calculating barycentric coord vectors
  double u = dot_vec3(o_a_vector, *d_cross_c_a) * inv_det; // u = s dot h * f
  // since the vectors are normalized, anything < 0 or > 1 means that the intersection
  // is not in the bounds of the triangle 
  if (u < 0.0 || u > 1.0){
    return false;
  }
  struct Vec3 * o_a_cross_b_a = cross_vec3(o_a_vector, b_a_vector); // q = s cross edge1
  double v = dot_vec3(*o_a_cross_b_a, ray->d) * inv_det;
  if (v < 0.0 || (v+u) > 1.0){
    free(o_a_cross_b_a);
    free(d_cross_c_a);
    return false;
  }
  double t = dot_vec3(*o_a_cross_b_a, c_a_vector) * inv_det;
  if (t > epsilon){
    *intersection_point = ray->o + (ray->d * t);
    free(o_a_cross_b_a);
    free(d_cross_c_a);
    return true;
  }
  free(o_a_cross_b_a);
  free(d_cross_c_a);
  return false;
}

// Update both or find a macro trick
#define FILE_LIST {"pyramid.stl"}//,"sphere.stl"}
#define NUMBER_OF_FILES 1
struct Vec3 file_offsets[NUMBER_OF_FILES] = {Vec3(0,0,100)};//, Vec3(100,0,0)};
#define DEBUG_MODE true

#define H 500 // pixel height
#define W 500 // pixel width
#define BRIGHTNESS 0.5
#define SCALING 5.0
#define OFFSET 0.0
#define ZOOM 1

#define STEPS 10

// generate a raytraced framed
// requires an array of stls, the number of stls, the output file name, the light angle (angle of the light source, this is ABSOLUTE)
// and the angle of the object (this is INCREMENTING, each frame generation with a given object angle MODIFIES THE STL)
__global__ void raytrace(struct STL *stl[], const int number_of_stls, Vec3 *output, float light_angle, float object_angle)
{
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int j = threadIdx.y + blockIdx.y * blockDim.y;
  if((i >= W) || (j >= H)) return;
  int pixel_index = j*W + i;

  // creating light source point
  double light_source_x = W/2+W*cos(light_angle)/2;
  double light_source_y = H/2+H*sin(light_angle)/2;
  double light_source_z = 500.0;
  const Sphere light(Vec3(light_source_x,light_source_y,light_source_z ), 1);

  const Vec3 white(MAX_PIXEL, MAX_PIXEL, MAX_PIXEL); // the red will likely need to substituted with surface parameters
  const Vec3 black(0, 0, 0);
  const Vec3 red(MAX_PIXEL, 0, 0);

  Vec3 pix_col(black);
  Vec3 *pi = (Vec3 *)hipMalloc(sizeof(Vec3));
  
  Vec3 pix_col_tmp = black;

  rotate_stl(ROT_Z, stl[0], object_angle);
  rotate_stl(ROT_X, stl[0], object_angle);
  rotate_stl(ROT_Y, stl[0], -object_angle);
  
  
  for (int z = 0; z < number_of_stls; z++) {
    pix_col = black;
    Ray ray(Vec3(i/ZOOM,j/ZOOM,0), Vec3(0,0,1));
    for (int ind = 0; ind < stl[z]->length; ind++){
      if(ray_triangle_intersect(&ray, &(stl[z]->triangles[ind]), pi)){
          const Vec3 L = light.c - *pi;
          const Vec3 N = stl[z]->triangles[ind].normal;
          const double dt = dot_vec3(L.normalize(), N.normalize());
          pix_col = (red + white*dt) * BRIGHTNESS;
          clamp_pixels(pix_col); 
          if(ind > 0){
            pix_col = pix_col_tmp.max(pix_col);
          }
          pix_col_tmp = pix_col;
      }
    }
    pix_col_tmp = black;
    output[pixel_index] = pix_col;
    hipFree(pi);
  }
  //   // debugging highlighting origin with red square
  //   if (i <= 10 && j <= 10 ){
  //     pix_col = Vec3(MAX_PIXEL,0,0);
  //   } else if (fabs(i - light_source_x) <= 1 && fabs(j-light_source_y ) <= 1){
  //     pix_col = white;
  //   }
  //   // paint y axis green
  //   if (j == 0){
  //     pix_col = Vec3(0,MAX_PIXEL,0);
  //   // paint x axis blue
  //   } else if (i == 0){
  //     pix_col = Vec3(0,0,MAX_PIXEL);
  //   }
  // }
  // for (int z = 0; z<number_of_stls; z++){

  //   // for (int i = 0; i < stl[z]->length; i++){
  //   //   struct Triangle * tri = &(stl[z]->triangles[i]); 
  //   //   free(tri);
  //   // }
  //   free(stl[z]);
  // }
  
}


// using Möller-Trumbore algorithm for raytracing w/ triangles 
int main() 
{
  struct STL *stl[NUMBER_OF_FILES];
  struct STL *objects[NUMBER_OF_FILES];
  const std::string filenames[NUMBER_OF_FILES] = FILE_LIST;

  double start_time, finish_time, total_time, current_time, increment_point;
  
  struct Parameters params = Parameters(SCALING, OFFSET, H, W);

  for (int i = 0; i < NUMBER_OF_FILES; i++) {
    stl[i] = load_stl(filenames[i], params, file_offsets[i]);
    std::cout << "Successfully loaded " <<  filenames[i] << "%s\n";
    printf("Number of triangles: %i\n", stl[i]->length);
  }

  std::string output_filename = "output/out.ppm";
  const int start = 0;
  start_time = CLOCK();
  increment_point = start_time;
  for (int i = start; i < STEPS+start; i++){
    std::string appended_info = std::to_string(i+1);

    // copy values to the gpu kernel
    uint32_t stl_size = NUMBER_OF_FILES * sizeof(struct STL);
    uint32_t output_size = H*W*sizeof(struct Vec3);
    Vec3 *output_values[H*W] = malloc(output_size);

    struct STL *stl_d = hipMalloc(stl_size);
    hipMemcpy(stl, stl_d, stl_size, hipMemcpyHostToDevice);
    struct Vec3 output_values_d = hipMalloc(output_size);
    hipMemcpy(output_values, output_values_d, output_size, hipMemcpyHostToDevice);


    raytrace<<H, W>>(stl_d, NUMBER_OF_FILES, output_values_d, i*2*M_PI/(float)STEPS, M_PI/(float)STEPS);
    //raytrace(stl, NUMBER_OF_FILES, i*2*M_PI/(float)STEPS,M_PI/(float)STEPS);

    // copy values back out
    hipMemcpy(output_values_d, output_values, output_size, hipMemcpyDeviceToHost);
    hipFree(output_values_d);
    hipFree(stl_d);

    // Save values locally
    output_filename.insert(10,appended_info);
    std::ofstream out(output_filename);
    out << "P3\n" << W << ' ' << H << ' ' << MAX_PIXEL<<"\n";

    for (int i = 0; i < H*W; i++) {
      out << (int) output_values[i].x << ' '
          << (int) output_values[i].y << ' '
          << (int) output_values[i].z << '\n';
    }
    out.close();
    
    output_filename = "output/out.ppm";
    if (DEBUG_MODE) {
      current_time = CLOCK();
      printf("Frame %d processed in %f ms\n", i, current_time-increment_point);
      increment_point = current_time;
    }
  }
  if (DEBUG_MODE) {
    finish_time = CLOCK();
    total_time = finish_time-start_time;
    printf("The total time to raytrace was: %f ms\n", total_time);
  }
}
