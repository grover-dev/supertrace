#include "hip/hip_runtime.h"
#include <fstream>
#include <stdlib.h>
#include <string>
#include <iostream>
#include <cmath>
#include <time.h>
#include <stdint.h>
#include "shapes.cuh"

#define FILE_LIST {"pyramid.stl"}
#define NUMBER_OF_FILES 1
// vector below applies variable offset to model center
struct Vec3 file_offsets[NUMBER_OF_FILES] = {Vec3(0,0,500)}; 
#define DEBUG_MODE true

#define H 500 // pixel height
#define W 500 // pixel width
#define BRIGHTNESS 0.5 //pixel brightness (0.5 = 100%)
#define SCALING 4.0 // object scaling
#define OFFSET 0.0 // object offset 
#define ZOOM 1

#define LIGHT_SOURCE_HEIGHT 1000

#define STEPS 10
// #define MPI

// clock for timing functions
__host__ double CLOCK() {
        struct timespec t;
        clock_gettime(CLOCK_MONOTONIC,  &t);
        return (t.tv_sec * 1000)+(t.tv_nsec*1e-6);
}

// pixel value clamping to be within defined ranges
#define MAX_PIXEL 1023
__device__ void clamp_pixels(Vec3& col)
{
  col.x = (col.x > MAX_PIXEL) ? MAX_PIXEL : (col.x < 0) ? 0 : col.x;
  col.y = (col.y > MAX_PIXEL) ? MAX_PIXEL : (col.y < 0) ? 0 : col.y;
  col.z = (col.z > MAX_PIXEL) ? MAX_PIXEL : (col.z < 0) ? 0 : col.z;
}

// used to calculate if ray intersects a given triangle
__device__ bool ray_triangle_intersect(struct Ray * ray, struct Triangle * tri, struct Vec3 * intersection_point){
  // error bound for 0
  bool intersect = true;
  const float epsilon = 0.00000001;

  struct Vec3 c_a_vector = tri->v2 - tri->v0; //edge2
  struct Vec3 b_a_vector = tri->v1 - tri->v0; //edge1 

  struct Vec3 d_cross_c_a = cross_vec3(ray->d, c_a_vector); 

  // first calculating determinant, 
  //  if its ~0 then the ray is parallel to the triangle
  //  if it is <0, then we are hitting the back of the triangle (counting as not intersecting for now)
  //  this will need to be adjusted in the future (especially with refraction) (i.e. use absolute value of det)
  // can therefore ignore it  
  double det = dot_vec3(d_cross_c_a, b_a_vector); 
  if (det < epsilon && det > -epsilon){
    // free(d_cross_c_a);
    return false;
  }
  double inv_det = 1.0 / det; // f = 1/a

  struct Vec3 o_a_vector = ray->o - tri->v0; // s = ray origin - vertex0

  // start calculating barycentric coord vectors
  double u = dot_vec3(o_a_vector, d_cross_c_a) * inv_det; 
  // since the vectors are normalized, anything < 0 or > 1 means that the intersection
  // is not in the bounds of the triangle 
  if (u < 0.0 || u > 1.0){
    intersect = intersect * 0;
  }
  struct Vec3 o_a_cross_b_a = cross_vec3(o_a_vector, b_a_vector); 
  double v = dot_vec3(o_a_cross_b_a, ray->d) * inv_det;
  if (v < 0.0 || (v+u) > 1.0){
    intersect = intersect * 0;
  }
  double t = dot_vec3(o_a_cross_b_a, c_a_vector) * inv_det;
  if (t > epsilon){
    *intersection_point = ray->o + (ray->d * t);
    return intersect;
  }
  return false;
}


// generate a raytraced framed
// requires an array of stls, the number of stls, the output pointer, the light angle (angle of the light source, this is ABSOLUTE)
// and the angle of the object (this is INCREMENTING, each frame generation with a given object angle MODIFIES THE STL)
__global__ void raytrace(struct STL *stl[], struct Triangle * tri_d, const int number_of_stls, Vec3 *output, float light_angle)
{
  int i = blockIdx.x ;
  int j = threadIdx.x ;
  if((i >= W) || (j >= H)) return;
  int pixel_index = j + i* blockDim.x;

  // creating light source point
  double light_source_x = W/2+W*cos(light_angle)/2;
  double light_source_y = H/2+H*sin(light_angle)/2;
  double light_source_z = LIGHT_SOURCE_HEIGHT;
  const Sphere light(Vec3(light_source_x,light_source_y,light_source_z ), 1);

  const struct Vec3 white(MAX_PIXEL, MAX_PIXEL, MAX_PIXEL); 
  const struct Vec3 black(0, 0, 0);
  const struct Vec3 red(MAX_PIXEL, 0, 0);

  struct Vec3 pix_col(black);
  struct Vec3 *pi;
  hipMalloc(&pi, sizeof(Vec3));
  
  for (int z = 0; z < number_of_stls; z++) {
    pix_col = black;
    Ray ray(Vec3(i/ZOOM,j/ZOOM,light_source_z/2), Vec3(0,0,1));
    for (int ind = 0; ind < stl[z]->length; ind++){
      if(ray_triangle_intersect(&ray, &(tri_d[ind]), pi)){
          const Vec3 L = light.c - *pi;
          const Vec3 N = stl[z]->triangles[ind].normal;
          const double dt = fabs(dot_vec3(L.normalize(), N.normalize()));
          pix_col = (red + white*dt) * BRIGHTNESS;
          clamp_pixels(pix_col);
      }
    }
    // debugging highlighting origin with red square
    if (i <= 10 && j <= 10 ){
      pix_col = Vec3(MAX_PIXEL,0,0);
    } else if (fabs(i - light_source_x) <= 1 && fabs(j-light_source_y ) <= 1){
      pix_col = white;
    }
    // paint y axis green
    if (j == 0){
      pix_col = Vec3(0,MAX_PIXEL,0);
    // paint x axis blue
    } else if (i == 0){
      pix_col = Vec3(0,0,MAX_PIXEL);
    }
    output[pixel_index] = pix_col;
  }
  hipFree(pi);  
}


// using Möller-Trumbore algorithm for raytracing w/ triangles 
void stl_raytracer_main(int frame_arr [], int frame_arr_length, int total_steps) 
{
  struct STL *stl[NUMBER_OF_FILES];
  const std::string filenames[NUMBER_OF_FILES] = FILE_LIST;

  double start_time, finish_time, total_time, current_time, increment_point;
  
  struct Parameters params = Parameters(SCALING, OFFSET, H, W);

  for (int i = 0; i < NUMBER_OF_FILES; i++) {
    stl[i] = load_stl(filenames[i], params, file_offsets[i]);
    std::cout << "Successfully loaded " <<  filenames[i] << "%s\n";
    printf("Number of triangles: %i\n", stl[i]->length);
  }

  std::string output_filename = "output/out.ppm";
  start_time = CLOCK();
  increment_point = start_time;

  hipError_t code;

  uint32_t stl_size = NUMBER_OF_FILES * (sizeof(struct STL));
  uint32_t output_size = H*W*sizeof(struct Vec3);
  Vec3 *output_values;
  code = hipHostMalloc(&output_values, output_size);
  struct STL **stl_d;
  struct Triangle * tri_d;
  uint32_t tri_size = sizeof(struct Triangle) * stl[0]->length;

  struct Vec3 *output_values_d;
  
  size_t free, total;
  hipMemGetInfo(&free,&total);
  printf("%d KB free of total %d KB\n",free/1024,total/1024);

  int last_frame = 0;
  for (int i = 0; i < frame_arr_length; i++){
    std::string appended_info = std::to_string(frame_arr[i]);
    

    float object_angle;
    if (i > 0){
      object_angle =  (frame_arr[i] - last_frame) * M_PI/(float)total_steps;
    } else {
      object_angle = frame_arr[i ] * M_PI/(float)total_steps; 
    }
    last_frame = frame_arr[i];


    // ADJUST THE ROTATION 
    rotate_stl(ROT_Z, stl[0], object_angle);
    rotate_stl(ROT_X, stl[0], object_angle);
    rotate_stl(ROT_Y, stl[0], object_angle);

    // copy values to the gpu kernel
    code = hipMalloc(&stl_d, stl_size);
    code = hipMemcpy(stl_d, stl, stl_size, hipMemcpyHostToDevice);

    code = hipMalloc(&tri_d, tri_size);
    code = hipMemcpy(tri_d, stl[0]->triangles, tri_size, hipMemcpyHostToDevice);


    // allocate output array
    code = hipMalloc(&output_values_d, output_size);
    code = hipMemcpy(output_values_d, output_values, output_size, hipMemcpyHostToDevice);

    // execute cuda raytracing
    raytrace<<<H, W>>>(stl_d, tri_d, NUMBER_OF_FILES, output_values_d, frame_arr[i]*M_PI/(float)total_steps);
    
    hipDeviceSynchronize();
    // copy values back out
    code = hipMemcpy(output_values, output_values_d, output_size, hipMemcpyDeviceToHost);

    // Save values locally
    output_filename.insert(10,appended_info);
    std::ofstream out(output_filename);
    out << "P3\n" << W << ' ' << H << ' ' << MAX_PIXEL<<"\n";

    for (int j = 0; j < H*W; j++) {
      out << (int) output_values[j].x << ' '
          << (int) output_values[j].y << ' '
          << (int) output_values[j].z << '\n';
    }
    out.close();
    
    output_filename = "output/out.ppm";
    if (DEBUG_MODE) {
      current_time = CLOCK();
      printf("Frame %d processed in %f ms\n", frame_arr[i], current_time-increment_point);
      increment_point = current_time;
    }
  }
  code = hipFree(output_values_d);
  code = hipFree(stl_d);
  if (DEBUG_MODE) {
    finish_time = CLOCK();
    total_time = finish_time-start_time;
    printf("The total time to raytrace was: %f ms\n", total_time);
  }
}

#ifdef MPI
  #include "mpi.h"
#endif
int main(int argc, char *argv[]){
  #ifdef MPI
    int numprocs, rank, namelen;
    char processor_name[MPI_MAX_PROCESSOR_NAME];

    // mpi initialization 
    MPI_Init(&argc, &argv);
    MPI_Comm_size(MPI_COMM_WORLD, &numprocs);
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Get_processor_name(processor_name, &namelen);
    int block_size = STEPS/numprocs;
  #else
    int block_size = STEPS;
    int rank = 0;
  #endif
  int * frame_arr = (int *)malloc(sizeof(int)*block_size);
  
  for (int i = 0; i< block_size; i++){
    frame_arr[i] = block_size * rank + i;
  }

  stl_raytracer_main(frame_arr, block_size, STEPS);
  #ifdef MPI
    MPI_Finalize();
  #endif
}