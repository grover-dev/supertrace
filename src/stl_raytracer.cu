#include "hip/hip_runtime.h"
#include <fstream>
#include <stdlib.h>
#include "shapes.cuh"
#include <string>
#include <iostream>
#include <cmath>
#include <time.h>
#include <stdint.h>


__host__ double CLOCK() {
        struct timespec t;
        clock_gettime(CLOCK_MONOTONIC,  &t);
        return (t.tv_sec * 1000)+(t.tv_nsec*1e-6);
}

#define MAX_PIXEL 1023
__device__ void clamp_pixels(Vec3& col)
{
  col.x = (col.x > MAX_PIXEL) ? MAX_PIXEL : (col.x < 0) ? 0 : col.x;
  col.y = (col.y > MAX_PIXEL) ? MAX_PIXEL : (col.y < 0) ? 0 : col.y;
  col.z = (col.z > MAX_PIXEL) ? MAX_PIXEL : (col.z < 0) ? 0 : col.z;
}



__device__ bool ray_triangle_intersect(struct Ray * ray, struct Triangle * tri, struct Vec3 * intersection_point){
  // error bound for 0
  const float epsilon = 0.00000001;

  struct Vec3 c_a_vector = tri->v2 - tri->v0; //edge2
  struct Vec3 b_a_vector = tri->v1 - tri->v0; //edge1 

  struct Vec3 d_cross_c_a = cross_vec3(ray->d, c_a_vector); // h = ray cross edge2

  // first calculating determinant, 
  //  if its ~0 then the ray is parallel to the triangle
  //  if it is <0, then we are hitting the back of the triangle (counting as not intersecting for now)
  //  this will need to be adjusted in the future (especially with refraction) (i.e. use absolute value of det)
  // can therefore ignore it  
  double det = dot_vec3(d_cross_c_a, b_a_vector); // a = (ray cross edge2) => h dot edge1
  if (det < epsilon && det > -epsilon){
    // free(d_cross_c_a);
    return false;
  }
  double inv_det = 1.0 / det; // f = 1/a

  struct Vec3 o_a_vector = ray->o - tri->v0; // s = ray origin - vertex0

  // start calculating barycentric coord vectors
  double u = dot_vec3(o_a_vector, d_cross_c_a) * inv_det; // u = s dot h * f
  // since the vectors are normalized, anything < 0 or > 1 means that the intersection
  // is not in the bounds of the triangle 
  if (u < 0.0 || u > 1.0){
    return false;
  }
  struct Vec3 o_a_cross_b_a = cross_vec3(o_a_vector, b_a_vector); // q = s cross edge1
  double v = dot_vec3(o_a_cross_b_a, ray->d) * inv_det;
  if (v < 0.0 || (v+u) > 1.0){
    // free(o_a_cross_b_a);
    // free(d_cross_c_a);
    return false;
  }
  double t = dot_vec3(o_a_cross_b_a, c_a_vector) * inv_det;
  if (t > epsilon){
    *intersection_point = ray->o + (ray->d * t);
    // free(o_a_cross_b_a);
    // free(d_cross_c_a);
    return true;
  }
  // free(o_a_cross_b_a);
  // free(d_cross_c_a);
  return false;
}

// Update both or find a macro trick
#define FILE_LIST {"pyramid.stl"}//,"sphere.stl"}
#define NUMBER_OF_FILES 1
struct Vec3 file_offsets[NUMBER_OF_FILES] = {Vec3(0,0,100)};//, Vec3(100,0,0)};
#define DEBUG_MODE true

#define H 500 // pixel height
#define W 500 // pixel width
#define BRIGHTNESS 0.5
#define SCALING 8.0
#define OFFSET 0.0
#define ZOOM 1

#define STEPS 20

// generate a raytraced framed
// requires an array of stls, the number of stls, the output file name, the light angle (angle of the light source, this is ABSOLUTE)
// and the angle of the object (this is INCREMENTING, each frame generation with a given object angle MODIFIES THE STL)
__global__ void raytrace(struct STL *stl[], const int number_of_stls, Vec3 *output, float light_angle, float object_angle)
{
  int i = blockIdx.x ;
  int j = threadIdx.x ;
  if((i >= W) || (j >= H)) return;
  int pixel_index = j + i* blockDim.x;

  // creating light source point
  double light_source_x = W/2+W*cos(light_angle)/2;
  double light_source_y = H/2+H*sin(light_angle)/2;
  double light_source_z = 5000.0;
  const Sphere light(Vec3(light_source_x,light_source_y,light_source_z ), 1);

  const struct Vec3 white(MAX_PIXEL, MAX_PIXEL, MAX_PIXEL); // the red will likely need to substituted with surface parameters
  const struct Vec3 black(0, 0, 0);
  const struct Vec3 red(MAX_PIXEL, 0, 0);

  struct Vec3 pix_col(black);
  struct Vec3 *pi;
  hipMalloc(&pi, sizeof(Vec3));
  // printf("stl point %lu\n", &stl[0]->triangles);
  // struct Vec3 tmp = stl[0]->triangles[0].v0;
  // printf("here_cuda, i: %i, j: %i\n",i,j);
  // printf("here_Cuda i: %i, j: %i, threadid: %i, blockid: %i, blockdim: %i\n Vec3: x = %f, y = %f, z=%f \n",
  //         i, j,threadIdx.x, blockIdx.x,blockDim.x , tmp.x,tmp.y,tmp.z);
  // printf("here_Cuda pixel: %i, i: %i, j: %i,  threadid: %i, blockid: %i, blockdim: %i\n",
  //         pixel_index, i, j, threadIdx.x, blockIdx.x,blockDim.x);


  
  for (int z = 0; z < number_of_stls; z++) {
    pix_col = black;
    Ray ray(Vec3(i/ZOOM,j/ZOOM,-100), Vec3(0,0,1));
    for (int ind = 0; ind < stl[z]->length; ind++){
      if(ray_triangle_intersect(&ray, &(stl[z]->triangles[ind]), pi)){
          const Vec3 L = light.c - *pi;
          const Vec3 N = stl[z]->triangles[ind].normal;
          const double dt = dot_vec3(L.normalize(), N.normalize());
          pix_col = (red + white*dt) * BRIGHTNESS;
          clamp_pixels(pix_col); 
      }
    }
    // debugging highlighting origin with red square
    if (i <= 10 && j <= 10 ){
      pix_col = Vec3(MAX_PIXEL,0,0);
    } else if (fabs(i - light_source_x) <= 1 && fabs(j-light_source_y ) <= 1){
      pix_col = white;
    }
    // paint y axis green
    if (j == 0){
      pix_col = Vec3(0,MAX_PIXEL,0);
    // paint x axis blue
    } else if (i == 0){
      pix_col = Vec3(0,0,MAX_PIXEL);
    }
    output[pixel_index] = pix_col;
  }
  hipFree(pi);
    
  // for (int z = 0; z<number_of_stls; z++){

  //   // for (int i = 0; i < stl[z]->length; i++){
  //   //   struct Triangle * tri = &(stl[z]->triangles[i]); 
  //   //   free(tri);
  //   // }
  //   free(stl[z]);
  // }
  
}


// using Möller-Trumbore algorithm for raytracing w/ triangles 
int main() 
{
  struct STL *stl[NUMBER_OF_FILES];
  const std::string filenames[NUMBER_OF_FILES] = FILE_LIST;

  double start_time, finish_time, total_time, current_time, increment_point;
  
  struct Parameters params = Parameters(SCALING, OFFSET, H, W);

  for (int i = 0; i < NUMBER_OF_FILES; i++) {
    stl[i] = load_stl(filenames[i], params, file_offsets[i]);
    std::cout << "Successfully loaded " <<  filenames[i] << "%s\n";
    printf("Number of triangles: %i\n", stl[i]->length);
  }

  std::string output_filename = "output/out.ppm";
  const int start = 0;
  start_time = CLOCK();
  increment_point = start_time;

  hipError_t code;

  uint32_t stl_size = NUMBER_OF_FILES * (stl[0]->length * sizeof(struct Triangle) + sizeof(uint32_t) + sizeof(struct Vec3));
  uint32_t output_size = H*W*sizeof(struct Vec3);
  Vec3 *output_values;
  code = hipHostMalloc(&output_values, output_size);
  struct STL **stl_d;
  struct Vec3 *output_values_d;
  
  for (int i = start; i < STEPS+start; i++){
    std::string appended_info = std::to_string(i+1);
    // copy values to the gpu kernel

    float object_angle =  M_PI/(float)STEPS;
  
    rotate_stl(ROT_Z, stl[0], object_angle);
    rotate_stl(ROT_X, stl[0], object_angle);
    // rotate_stl(ROT_Y, stl[0], -object_angle);

    code = hipMalloc(&stl_d, stl_size);
    code = hipMemcpy(stl_d, stl, stl_size, hipMemcpyHostToDevice);
    code = hipMalloc(&output_values_d, output_size);
    code = hipMemcpy(output_values_d, output_values, output_size, hipMemcpyHostToDevice);


    raytrace<<<H, W>>>(stl_d, NUMBER_OF_FILES, output_values_d, i*M_PI/(float)STEPS, M_PI/(float)STEPS);
    //raytrace(stl, NUMBER_OF_FILES, i*2*M_PI/(float)STEPS,M_PI/(float)STEPS);

    // copy values back out
    code = hipMemcpy(output_values, output_values_d, output_size, hipMemcpyDeviceToHost);
    code = hipFree(output_values_d);
    code = hipFree(stl_d);

    // Save values locally
    output_filename.insert(10,appended_info);
    std::ofstream out(output_filename);
    out << "P3\n" << W << ' ' << H << ' ' << MAX_PIXEL<<"\n";

    for (int j = 0; j < H*W; j++) {
      out << (int) output_values[j].x << ' '
          << (int) output_values[j].y << ' '
          << (int) output_values[j].z << '\n';
    }
    out.close();
    
    output_filename = "output/out.ppm";
    if (DEBUG_MODE) {
      current_time = CLOCK();
      printf("Frame %d processed in %f ms\n", i, current_time-increment_point);
      increment_point = current_time;
    }
  }
  if (DEBUG_MODE) {
    finish_time = CLOCK();
    total_time = finish_time-start_time;
    printf("The total time to raytrace was: %f ms\n", total_time);
  }
}
